#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <stdint.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_math_constants.h>

// Keep this in sync with optix_wrapper.cpp
struct Params
{
  hipDeviceptr_t              out_rgb;      // float3*
  hipDeviceptr_t              out_bayer;    // float*
  int                      width;
  int                      height;
  int                      spp;
  int                      max_depth;
  int                      frame;
  int                      bayer_pattern; // 0:RGGB,1:BGGR,2:GRBG,3:GBRG
  float3                   cam_eye;
  float3                   cam_u;
  float3                   cam_v;
  float3                   cam_w;
  OptixTraversableHandle   handle;

  // Geometry arrays
  hipDeviceptr_t              d_vertices;   // float3*
  hipDeviceptr_t              d_indices;    // uint3*
  hipDeviceptr_t              d_normals;    // float3*
  uint32_t                 num_triangles;

  // Rectangular area light centered at light_pos with half extents light_half
  float3                   light_pos;
  float3                   light_emit;
  float3                   light_normal;
  float2                   light_half;
};

extern "C" {
__constant__ Params params;
}

// SBT data for each primitive
struct HitgroupData
{
  float3 Base_colour;
  float  Metallic;
  float  Roughness;
  float  IOR;
  float  Alpha;
  float  Transmission;
  float3 Emission;
};

enum {
    RAY_TYPE_RADIANCE = 0,
    RAY_TYPE_SHADOW   = 1,
    RAY_TYPE_COUNT    = 2
};

// --- float3 helpers (device-only) -----------------------------------------

static __forceinline__ __device__ float3 make3(float s) { return make_float3(s,s,s); }

static __forceinline__ __device__ float3 operator+(const float3& a, const float3& b) {
  return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}
static __forceinline__ __device__ float3 operator-(const float3& a, const float3& b) {
  return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}
static __forceinline __device__ float3 operator-(const float3& a) {
    return make_float3(-a.x, -a.y, -a.z);
}
static __forceinline__ __device__ float3 operator*(const float3& a, float s) {
  return make_float3(a.x*s, a.y*s, a.z*s);
}
static __forceinline__ __device__ float3 operator*(float s, const float3& a) {
  return make_float3(a.x*s, a.y*s, a.z*s);
}
static __forceinline__ __device__ float3 operator*(const float3& a, const float3& b) {
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}
static __forceinline__ __device__ float3 mul(const float3& a, const float3& b) {
  return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);
}
static __forceinline__ __device__ float3 operator/(const float3& a, float s) {
  float inv = 1.0f/s;
  return make_float3(a.x*inv, a.y*inv, a.z*inv);
}
static __forceinline__ __device__ float3& operator+=(float3& a, const float3& b) {
    a.x += b.x; a.y += b.y; a.z += b.z; return a;
}
static __forceinline__ __device__ float dot(const float3& a, const float3& b) {
    return a.x*b.x + a.y*b.y + a.z*b.z;
}
static __forceinline__ __device__ float3 cross(const float3& a, const float3& b) {
    return make_float3(a.y*b.z - a.z*b.y,
                       a.z*b.x - a.x*b.z,
                       a.x*b.y - a.y*b.x);
}
static __forceinline__ __device__ float length(const float3& v) {
    return sqrtf(dot(v, v));
}
static __forceinline__ __device__ float3 normalize(const float3& v) {
    const float len = length(v);
    return (len > 0.f) ? v / len : make_float3(0.f, 0.f, 0.f);
}
static __forceinline__ __device__ float  dot3(const float3& a, const float3& b) {
  return a.x*b.x + a.y*b.y + a.z*b.z;
}
static __forceinline__ __device__ float3 cross3(const float3& a, const float3& b) {
  return make_float3(a.y*b.z - a.z*b.y,
                     a.z*b.x - a.x*b.z,
                     a.x*b.y - a.y*b.x);
}
static __forceinline__ __device__ float  length3(const float3& a) {
  return sqrtf(dot3(a,a));
}
static __forceinline__ __device__ float3 normalize3(const float3& a) {
  return a / fmaxf(length3(a), 1e-20f);
}
static __forceinline__ __device__ float  clamp01(float x) {
  return fminf(fmaxf(x, 0.0f), 1.0f);
}

static __forceinline__ __device__ float select(const float3& v, int ch) {
  return ch==0 ? v.x : (ch==1 ? v.y : v.z);
}
static __forceinline__ __device__ unsigned int pcg(unsigned long long& state) {
  unsigned long long oldstate = state;
  state = oldstate * 6364136223846793005ULL + 1442695040888963407ULL;
  unsigned int xorshifted = ((oldstate >> 18u) ^ oldstate) >> 27u;
  unsigned int rot = oldstate >> 59u;
  return (xorshifted >> rot) | (xorshifted << ((-rot) & 31));
}

static __forceinline__ __device__ float rnd(unsigned long long& state) {
  return float(pcg(state) >> 8) * (1.0f / float(0x01000000u));
}

// Generate spatiotemporal blue noise sample using an R2 sequence with a
// per-pixel Cranley-Patterson rotation. The hash introduces a high frequency
// offset so neighbouring pixels are decorrelated while preserving the low
// discrepancy of the base sequence.
static __forceinline__ __device__ unsigned int hash_u32(unsigned int x)
{
  x ^= x >> 17;
  x *= 0xed5ad4bbU;
  x ^= x >> 11;
  x *= 0xac4c1b51U;
  x ^= x >> 15;
  x *= 0x31848babU;
  x ^= x >> 14;
  return x;
}

static __forceinline__ __device__ float2 blue_noise(int x, int y, int s, int frame)
{
  const float a1 = 1.0f / 1.32471795724474602596f;      // plastic constant
  const float a2 = 1.0f / (1.32471795724474602596f * 1.32471795724474602596f);
  unsigned int h = hash_u32(x * 1973u + y * 9277u + frame * 26699u);
  float rx = (h & 0xffff) * (1.0f / 65536.0f);
  float ry = ((h >> 16) & 0xffff) * (1.0f / 65536.0f);
  float u = fmodf(rx + (s + 0.5f) * a1, 1.0f);
  float v = fmodf(ry + (s + 0.5f) * a2, 1.0f);
  return make_float2(u, v);
}

static __forceinline__ __device__ float sample_tri(float u)
{
  return (u < 0.5f) ? sqrtf(2.0f * u) - 1.0f : 1.0f - sqrtf(2.0f * (1.0f - u));
}

static __forceinline__ __device__ float2 olpf_jitter(int x, int y, int s, int frame)
{
  float2 u = blue_noise(x, y, s, frame);
  return make_float2(sample_tri(u.x), sample_tri(u.y));
}

static __forceinline__ __device__ void make_onb(const float3& n, float3& t, float3& b)
{
  if (n.z < -0.9999999f) {
    t = make_float3(0.0f, -1.0f, 0.0f);
    b = make_float3(-1.0f, 0.0f, 0.0f);
  } else {
    const float a = 1.0f / (1.0f + n.z);
    const float naxy = -n.x * n.y * a;
    t = make_float3(1.0f - n.x * n.x * a, naxy, -n.x);
    b = make_float3(naxy, 1.0f - n.y * n.y * a, -n.y);
  }
}

static __forceinline__ __device__ float3 reflect(const float3& i, const float3& n)
{
  return i - 2.0f * dot(i, n) * n;
}

static __forceinline__ __device__ float3 fresnel_schlick(float cosTheta, const float3& F0)
{
  return F0 + (make3(1.0f) - F0) * powf(fmaxf(0.0f, 1.0f - cosTheta), 5.0f);
}

// Sample the GGX microfacet distribution using the Visible Normal Distribution
// Function (VNDF) technique. This method better matches the distribution of
// visible microfacets for the given view direction, reducing variance compared
// to sampling the full NDF. See "Sampling the GGX Distribution of Visible
// Normals" by Heitz (2018).
static __forceinline__ __device__ void sample_ggx_vndf(
    float2 u, float alpha, const float3& N, const float3& V, float3& L, float3& H, float& pdf)
{
  // Build an orthonormal basis and transform the view direction to local space
  float3 T, B;
  make_onb(N, T, B);
  float3 Vlocal = make_float3(dot(V, T), dot(V, B), dot(V, N));

  // Stretch view direction by the roughness parameter (Heitz 2018)
  float3 Vh = normalize(make_float3(alpha * Vlocal.x, alpha * Vlocal.y, Vlocal.z));

  // Orthonormal basis around the stretched view direction
  float lensq = Vh.x * Vh.x + Vh.y * Vh.y;
  float3 T1 = lensq > 0.0f ? make_float3(-Vh.y, Vh.x, 0.0f) / sqrtf(lensq)
                           : make_float3(1.0f, 0.0f, 0.0f);
  float3 T2 = cross(Vh, T1);

  // Sample a point on a disk (polar coordinates)
  float r = sqrtf(u.x);
  float phi = 2.0f * HIP_PI_F * u.y;
  float t1 = r * cosf(phi);
  float t2 = r * sinf(phi);

  // Adjust t2 based on view direction to obtain a visible normal
  float s = 0.5f * (1.0f + Vh.z);
  t2 = (1.0f - s) * sqrtf(fmaxf(0.0f, 1.0f - t1 * t1)) + s * t2;

  // Reproject onto hemisphere and unstretch
  float3 Nh = t1 * T1 + t2 * T2 + sqrtf(fmaxf(0.0f, 1.0f - t1 * t1 - t2 * t2)) * Vh;
  float3 Hlocal = normalize(make_float3(alpha * Nh.x, alpha * Nh.y, fmaxf(0.0f, Nh.z)));

  // Transform the half-vector back to world space
  H = normalize(Hlocal.x * T + Hlocal.y * B + Hlocal.z * N);

  // Compute the reflected direction
  L = normalize(reflect(-V, H));

  // Compute PDF for the sampled direction. For VNDF sampling, this is
  // D(h) * G1(v) * (n·h) / (4 * (v·h))
  float NoH = fmaxf(dot(N, H), 0.0f);
  float NoV = fmaxf(dot(N, V), 0.0f);
  float VoH = fmaxf(dot(V, H), 0.0f);
  float alpha2 = alpha * alpha;
  float denom = NoH * NoH * (alpha2 - 1.0f) + 1.0f;
  float D = alpha2 / (HIP_PI_F * denom * denom);
  float G1V = 2.0f * NoV / (NoV + sqrtf(alpha2 + (1.0f - alpha2) * NoV * NoV));
  pdf = (D * NoH * G1V) / (4.0f * fmaxf(VoH, 1e-6f));
}

// --- payload packing -------------------------------------------------------

static __forceinline__ __device__ void packPtr(void* ptr, unsigned int& u0, unsigned int& u1) {
  uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
  u0 = static_cast<unsigned int>(uptr & 0xFFFFFFFFu);
  u1 = static_cast<unsigned int>(uptr >> 32);
}

template<typename T>
static __forceinline__ __device__ T* unpackPtr() {
  uint64_t u0 = static_cast<uint64_t>(optixGetPayload_0());
  uint64_t u1 = static_cast<uint64_t>(optixGetPayload_1());
  uint64_t uptr = (u1 << 32) | u0;
  return reinterpret_cast<T*>(uptr);
}

// --- per-ray data ----------------------------------------------------------

struct PRD {
    float3 radiance;
    float3 throughput;
    float3 origin;
    float3 direction;
    unsigned long long seed;
    int    depth;
    int    done;
    // MIS: previous bounce's BSDF pdf and validity flag
    float  prev_pdf_bsdf;
    int    prev_pdf_valid;
};

struct PRDScalar {
    float  radiance;
    float  throughput;
    float3 origin;
    float3 direction;
    unsigned long long seed;
    int    depth;
    int    done;
    float  prev_pdf_bsdf;
    int    prev_pdf_valid;
    int    ch;
};

// --- access helpers for hipDeviceptr_t arrays --------------------------------

template<typename T>
static __forceinline__ __device__ T* ptr_at(hipDeviceptr_t p) {
  return reinterpret_cast<T*>(p);
}

// --- Miss programs ---------------------------------------------------------

extern "C" __global__ void __miss__ms_radiance()
{
  PRD* prd = unpackPtr<PRD>();
  prd->radiance = make3(0.0f);
  prd->done = 1;
}

extern "C" __global__ void __miss__ms_radiance_scalar()
{
  PRDScalar* prd = unpackPtr<PRDScalar>();
  prd->radiance = 0.0f;
  prd->done = 1;
}

extern "C" __global__ void __miss__ms_shadow()
{
  // 0 means not occluded; anyhit will set to 1
  optixSetPayload_0(0u);
}

// --- Anyhit for shadow rays: mark occluded and terminate -------------------

extern "C" __global__ void __anyhit__ah_shadow()
{
    optixSetPayload_0(1u);
    optixTerminateRay();
}

// --- Closest hit for radiance rays ----------------------------------------

extern "C" __global__ void __closesthit__ch()
{
    PRD& prd = *unpackPtr<PRD>();

    const unsigned int prim = optixGetPrimitiveIndex();

    // Geometry from Params (NOT from SBT)
    const uint3*  indices  = reinterpret_cast<const uint3*>(params.d_indices);
    const float3* vertices = reinterpret_cast<const float3*>(params.d_vertices);
    const uint3 tri = indices[prim];

    const float3 v0 = vertices[tri.x];
    const float3 v1 = vertices[tri.y];
    const float3 v2 = vertices[tri.z];

    const float2 bc = optixGetTriangleBarycentrics();
    const float b1 = bc.x;
    const float b2 = bc.y;
    const float b0 = 1.0f - b1 - b2;

    const float3 P  = v0 * b0 + v1 * b1 + v2 * b2;
    const float3 Ng = normalize3(reinterpret_cast<const float3*>(params.d_normals)[prim]);

    // Fetch per-triangle materials from SBT
    const HitgroupData* hg = reinterpret_cast<const HitgroupData*>(optixGetSbtDataPointer());
    const float3 Base_colour = hg->Base_colour;
    const float Roughness = hg->Roughness;
    const float IOR = hg->IOR;
    const float3 Emission = hg->Emission;

    float f0 = (IOR - 1.0f) / (IOR + 1.0f);
    f0 = f0 * f0;
    float3 F0 = make3(f0);
    float spec_prob = fminf(0.99f, fmaxf(0.01f, (F0.x + F0.y + F0.z) * (1.0f / 3.0f)));
    float diff_prob = 1.0f - spec_prob;
    float3 kd = Base_colour * (1.0f - f0);

    // Sample direct illumination from rectangular area light
    unsigned long long& seed = prd.seed;
    float3 Lo = make3(0.0f);
    {
        float u = (rnd(seed) * 2.0f - 1.0f) * params.light_half.x;
        float v = (rnd(seed) * 2.0f - 1.0f) * params.light_half.y;
        float3 lp = make_float3(params.light_pos.x + u, params.light_pos.y + v, params.light_pos.z);
        float3 L = lp - P;
        float dist2 = fmaxf(dot(L, L), 1e-6f);
        float dist = sqrtf(dist2);
        float3 wi = L / dist;
        float3 light_n = make_float3(0.0f, 0.0f, -1.0f);
        float cosS = fmaxf(0.0f, dot(Ng, wi));
        float cosL = fmaxf(0.0f, dot(params.light_normal, wi * -1.0f));
        if (cosS > 0.0f && cosL > 0.0f) {
            unsigned int occluded = 0u;
            optixTrace(
                params.handle,
                P + Ng * 1e-3f,
                wi,
                0.0f,
                dist - 1e-3f,
                0.0f,
                1,
                OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
                RAY_TYPE_SHADOW,
                RAY_TYPE_COUNT,
                RAY_TYPE_SHADOW,
                occluded);
            if (!occluded) {
                float area = 4.0f * params.light_half.x * params.light_half.y;
                float pdf_area = 1.0f / area;
                float pdf_light = pdf_area * dist2 / cosL;
                float pdf_bsdf = cosS * (1.0f / HIP_PI_F) * diff_prob;
                float w = pdf_light / (pdf_light + pdf_bsdf);
                float3 f = kd * (1.0f / HIP_PI_F);
                float3 contrib = params.light_emit * f * (cosS * cosL / dist2) / pdf_area;
                Lo += contrib * w;
            }
        }
    }

    // Accumulate emission with MIS and direct light
    float3 emission = Emission;
    if (prd.prev_pdf_valid && (Emission.x > 0.0f || Emission.y > 0.0f || Emission.z > 0.0f)) {
        float3 L = P - prd.origin;
        float dist2 = fmaxf(dot(L, L), 1e-6f);
        float area = 4.0f * params.light_half.x * params.light_half.y;
        float3 light_n = make_float3(0.0f, 0.0f, -1.0f);
        float cosL = fmaxf(0.0f, dot(params.light_normal, prd.direction * -1.0f));
        float pdf_light = dist2 / (cosL * area);
        float w_bsdf = prd.prev_pdf_bsdf / (prd.prev_pdf_bsdf + pdf_light);
        emission = emission * w_bsdf;
    }
    prd.radiance += emission + Lo;

    float choose = rnd(seed);
    if (choose < spec_prob) {
        float2 u = make_float2(rnd(seed), rnd(seed));
        float3 V = -prd.direction;
        float3 newDir;
        float3 H;
        float pdf;
        float alpha = fmaxf(Roughness * Roughness, 1e-4f);
        sample_ggx_vndf(u, alpha, Ng, V, newDir, H, pdf);
        float NoL = fmaxf(dot(Ng, newDir), 0.0f);
        float NoV = fmaxf(dot(Ng, V), 0.0f);
        float VoH = fmaxf(dot(V, H), 0.0f);
        float NoH = fmaxf(dot(Ng, H), 0.0f);
        float alpha2 = alpha * alpha;
        float G1L = 2.0f * NoL /
                    (NoL + sqrtf(alpha2 + (1.0f - alpha2) * NoL * NoL));
        float3 F = fresnel_schlick(VoH, F0);
        float3 spec = F * (G1L * VoH / fmaxf(NoV * NoH, 1e-6f));
        prd.origin = P + Ng * 1e-3f;
        prd.direction = newDir;
        prd.throughput = mul(prd.throughput, spec / spec_prob);
        prd.prev_pdf_bsdf = pdf * spec_prob;
        prd.prev_pdf_valid = 1;
    } else {
        float r1 = rnd(seed);
        float r2 = rnd(seed);
        const float phi = 2.0f * HIP_PI_F * r1;
        const float cosTheta = sqrtf(1.0f - r2);
        const float sinTheta = sqrtf(r2);
        float3 localDir = make_float3(cosf(phi) * sinTheta, sinf(phi) * sinTheta, cosTheta);

        float3 tangent, bitangent;
        make_onb(Ng, tangent, bitangent);
        float3 newDir = normalize(localDir.x * tangent + localDir.y * bitangent + localDir.z * Ng);

        prd.origin = P + Ng * 1e-3f;
        prd.direction = newDir;
        prd.throughput = mul(prd.throughput, kd / diff_prob);
        prd.prev_pdf_bsdf = cosTheta * (1.0f / HIP_PI_F) * diff_prob;
        prd.prev_pdf_valid = 1;
    }

    prd.depth++;
    if (prd.depth >= params.max_depth) {
        prd.done = 1;
        return;
    }
    if (prd.depth >= params.max_depth - 2) {
        float p = fmaxf(prd.throughput.x, fmaxf(prd.throughput.y, prd.throughput.z));
        if (rnd(seed) > p) {
            prd.done = 1;
            return;
        }
        prd.throughput = prd.throughput / fmaxf(p, 1e-3f);
    }
}

extern "C" __global__ void __closesthit__ch_bayer()
{
    PRDScalar& prd = *unpackPtr<PRDScalar>();

    const unsigned int prim = optixGetPrimitiveIndex();

    const uint3*  indices  = reinterpret_cast<const uint3*>(params.d_indices);
    const float3* vertices = reinterpret_cast<const float3*>(params.d_vertices);
    const uint3 tri = indices[prim];

    const float3 v0 = vertices[tri.x];
    const float3 v1 = vertices[tri.y];
    const float3 v2 = vertices[tri.z];

    const float2 bc = optixGetTriangleBarycentrics();
    const float b1 = bc.x;
    const float b2 = bc.y;
    const float b0 = 1.0f - b1 - b2;

    const float3 P  = v0 * b0 + v1 * b1 + v2 * b2;
    const float3 Ng = normalize3(reinterpret_cast<const float3*>(params.d_normals)[prim]);

    const HitgroupData* hg = reinterpret_cast<const HitgroupData*>(optixGetSbtDataPointer());
    const float3 Base_colour = hg->Base_colour;
    const float Roughness = hg->Roughness;
    const float IOR = hg->IOR;
    const float3 Emission = hg->Emission;

    const int ch = prd.ch;
    float albedo = select(Base_colour, ch);
    float emission = select(Emission, ch);

    float f0 = (IOR - 1.0f) / (IOR + 1.0f);
    f0 = f0 * f0;
    float spec_prob = fminf(0.99f, fmaxf(0.01f, f0));
    float diff_prob = 1.0f - spec_prob;
    float kd = albedo * (1.0f - f0);

    unsigned long long& seed = prd.seed;
    float Lo = 0.0f;
    {
        float u = (rnd(seed) * 2.0f - 1.0f) * params.light_half.x;
        float v = (rnd(seed) * 2.0f - 1.0f) * params.light_half.y;
        float3 lp = make_float3(params.light_pos.x + u, params.light_pos.y + v, params.light_pos.z);
        float3 L = lp - P;
        float dist2 = fmaxf(dot(L, L), 1e-6f);
        float dist = sqrtf(dist2);
        float3 wi = L / dist;
        float cosS = fmaxf(0.0f, dot(Ng, wi));
        float cosL = fmaxf(0.0f, dot(params.light_normal, wi * -1.0f));
        if (cosS > 0.0f && cosL > 0.0f) {
            unsigned int occluded = 0u;
            optixTrace(
                params.handle,
                P + Ng * 1e-3f,
                wi,
                0.0f,
                dist - 1e-3f,
                0.0f,
                1,
                OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
                RAY_TYPE_SHADOW,
                RAY_TYPE_COUNT,
                RAY_TYPE_SHADOW,
                occluded);
            if (!occluded) {
                float area = 4.0f * params.light_half.x * params.light_half.y;
                float pdf_area = 1.0f / area;
                float pdf_light = pdf_area * dist2 / cosL;
                float pdf_bsdf = cosS * (1.0f / HIP_PI_F) * diff_prob;
                float w = pdf_light / (pdf_light + pdf_bsdf);
                float f = kd * (1.0f / HIP_PI_F);
                float Le = select(params.light_emit, ch);
                float contrib = Le * f * (cosS * cosL / dist2) / pdf_area;
                Lo += contrib * w;
            }
        }
    }

    if (prd.prev_pdf_valid && emission > 0.0f) {
        float3 L = P - prd.origin;
        float dist2 = fmaxf(dot(L, L), 1e-6f);
        float area = 4.0f * params.light_half.x * params.light_half.y;
        float cosL = fmaxf(0.0f, dot(params.light_normal, prd.direction * -1.0f));
        float pdf_light = dist2 / (cosL * area);
        float w_bsdf = prd.prev_pdf_bsdf / (prd.prev_pdf_bsdf + pdf_light);
        emission *= w_bsdf;
    }
    prd.radiance += emission + Lo;

    float choose = rnd(seed);
    if (choose < spec_prob) {
        float2 u = make_float2(rnd(seed), rnd(seed));
        float3 V = -prd.direction;
        float3 newDir;
        float3 H;
        float pdf;
        float alpha = fmaxf(Roughness * Roughness, 1e-4f);
        sample_ggx_vndf(u, alpha, Ng, V, newDir, H, pdf);
        float NoL = fmaxf(dot(Ng, newDir), 0.0f);
        float NoV = fmaxf(dot(Ng, V), 0.0f);
        float VoH = fmaxf(dot(V, H), 0.0f);
        float NoH = fmaxf(dot(Ng, H), 0.0f);
        float alpha2 = alpha * alpha;
        float G1L = 2.0f * NoL /
                    (NoL + sqrtf(alpha2 + (1.0f - alpha2) * NoL * NoL));
        float F = f0 + (1.0f - f0) * powf(fmaxf(0.0f, 1.0f - VoH), 5.0f);
        float spec = F * (G1L * VoH / fmaxf(NoV * NoH, 1e-6f));
        prd.origin = P + Ng * 1e-3f;
        prd.direction = newDir;
        prd.throughput *= spec / spec_prob;
        prd.prev_pdf_bsdf = pdf * spec_prob;
        prd.prev_pdf_valid = 1;
    } else {
        float r1 = rnd(seed);
        float r2 = rnd(seed);
        const float phi = 2.0f * HIP_PI_F * r1;
        const float cosTheta = sqrtf(1.0f - r2);
        const float sinTheta = sqrtf(r2);
        float3 localDir = make_float3(cosf(phi) * sinTheta, sinf(phi) * sinTheta, cosTheta);

        float3 tangent, bitangent;
        make_onb(Ng, tangent, bitangent);
        float3 newDir = normalize(localDir.x * tangent + localDir.y * bitangent + localDir.z * Ng);

        prd.origin = P + Ng * 1e-3f;
        prd.direction = newDir;
        prd.throughput *= kd / diff_prob;
        prd.prev_pdf_bsdf = cosTheta * (1.0f / HIP_PI_F) * diff_prob;
        prd.prev_pdf_valid = 1;
    }

    prd.depth++;
    if (prd.depth >= params.max_depth) {
        prd.done = 1;
        return;
    }
    if (prd.depth >= params.max_depth - 2) {
        float p = fminf(0.95f, fmaxf(0.05f, prd.throughput));
        if (rnd(seed) > p) {
            prd.done = 1;
            return;
        }
        prd.throughput *= (1.0f / p);
    }
}

// --- Raygen program --------------------------------------------------------

static __forceinline__ __device__ float3 sample_camera_dir(int x, int y, const float2& jitter)
{
  const uint3 dim = optixGetLaunchDimensions();
  float fx = (float(x) + 0.5f + jitter.x) / float(dim.x);
  float fy = (float(y) + 0.5f + jitter.y) / float(dim.y);
  fx = clamp01(fx);
  fy = clamp01(fy);
  const float2 d = make_float2(2.0f * fx - 1.0f, 1.0f - 2.0f * fy);
  return normalize3(params.cam_w + d.x * params.cam_u + d.y * params.cam_v);
}

static __forceinline__ __device__ int bayer_channel_for(int x, int y, int pattern)
{
  const int mx = x & 1;
  const int my = y & 1;
  switch(pattern){
    case 0: return (my==0) ? (mx==0 ? 0:1) : (mx==0 ? 1:2);
    case 1: return (my==0) ? (mx==0 ? 2:1) : (mx==0 ? 1:0);
    case 2: return (my==0) ? (mx==0 ? 1:0) : (mx==0 ? 2:1);
    default:return (my==0) ? (mx==0 ? 1:2) : (mx==0 ? 0:1);
  }
}

extern "C" __global__ void __raygen__rg()
{
  const uint3  idx = optixGetLaunchIndex();
  const int    x   = int(idx.x);
  const int    y   = int(idx.y);
  const int    W   = params.width;
  const int    H   = params.height;
  const int    dst = y * W + x;

  // Trace radiance
  const float3 org = params.cam_eye;
  float3 sum_rgb = make3(0.0f);
  float sum_bayer = 0.0f;
  const bool do_rgb   = params.out_rgb   != 0;
  const bool do_bayer = params.out_bayer != 0;
  const int ch = bayer_channel_for(x, y, params.bayer_pattern);
  unsigned long long seed =
      ((unsigned long long)params.frame * 9781ULL) ^
      ((unsigned long long)dst * 6271ULL) ^
      0x853c49e6748fea9bULL;

  for (int s = 0; s < params.spp; ++s) {
    PRD prd;
    prd.radiance = make3(0.0f);
    prd.throughput = make3(1.0f);
    prd.origin = org;
    float2 jitter = olpf_jitter(x, y, s, params.frame);
    prd.direction = sample_camera_dir(x, y, jitter);
    prd.depth = 0;
    prd.done = 0;
    prd.seed = seed;
    prd.prev_pdf_bsdf = 0.0f;
    prd.prev_pdf_valid = 0;

    while (!prd.done) {
      prd.radiance = make3(0.0f);
      unsigned int u0, u1; packPtr(&prd, u0, u1);
      optixTrace(
        params.handle,
        prd.origin,
        prd.direction,
        0.0f, 1e16f, 0.0f,
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_DISABLE_ANYHIT,
        0,
        2,
        0,
        u0, u1);
      if (do_rgb)
        sum_rgb += prd.radiance * prd.throughput;
      if (do_bayer) {
        float throughput_ch = (ch==0 ? prd.throughput.x : (ch==1 ? prd.throughput.y : prd.throughput.z));
        sum_bayer += (ch==0 ? prd.radiance.x : (ch==1 ? prd.radiance.y : prd.radiance.z)) * throughput_ch;
	  }
    }

    seed = prd.seed;
  }

  // Write outputs
  if (do_rgb) {
                float3 rad = sum_rgb / float(params.spp);
    float3* out = ptr_at<float3>(params.out_rgb);
    out[dst] = rad;
  }
  if (do_bayer) {
                float rad = fmaxf(sum_bayer / float(params.spp), 0.f);
    float* out = ptr_at<float>(params.out_bayer);
    out[dst] = rad;
  }
}

extern "C" __global__ void __raygen__bayer()
{
  const uint3  idx = optixGetLaunchIndex();
  const int    x   = int(idx.x);
  const int    y   = int(idx.y);
  const int    W   = params.width;
  const int    H   = params.height;
  const int    dst = y * W + x;

  const float3 org = params.cam_eye;
  float sum = 0.0f;
  const int ch = bayer_channel_for(x, y, params.bayer_pattern);
  unsigned long long seed =
      ((unsigned long long)params.frame * 9781ULL) ^
      ((unsigned long long)dst * 6271ULL) ^
      0x37c4d1e74c3fa19bULL;

  for (int s = 0; s < params.spp; ++s) {
    PRDScalar prd;
    prd.radiance = 0.0f;
    prd.throughput = 1.0f;
    prd.origin = org;
    float2 jitter = olpf_jitter(x, y, s, params.frame);
    prd.direction = sample_camera_dir(x, y, jitter);
    prd.depth = 0;
    prd.done = 0;
    prd.seed = seed;
    prd.prev_pdf_bsdf = 0.0f;
    prd.prev_pdf_valid = 0;
    prd.ch = ch;

    while (!prd.done) {
      prd.radiance = 0.0f;
      unsigned int u0, u1; packPtr(&prd, u0, u1);
      optixTrace(
        params.handle,
        prd.origin,
        prd.direction,
        0.0f, 1e16f, 0.0f,
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_DISABLE_ANYHIT,
        0,
        2,
        0,
        u0, u1);
      sum += prd.radiance * prd.throughput;
    }

    seed = prd.seed;
  }

  float rad = fmaxf(sum / float(params.spp), 0.f);
  float* out = ptr_at<float>(params.out_bayer);
  out[dst] = rad;
}
