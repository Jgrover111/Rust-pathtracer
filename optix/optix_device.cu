#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <stdint.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_math_constants.h>

// Keep this in sync with optix_wrapper.cpp
struct Params
{
  hipDeviceptr_t              out_rgb;      // float3*
  hipDeviceptr_t              out_bayer;    // float*
  int                      width;
  int                      height;
  int                      spp;
  int                      frame;
  int                      bayer_pattern; // 0:RGGB,1:BGGR,2:GRBG,3:GBRG
  float3                   cam_eye;
  float3                   cam_u;
  float3                   cam_v;
  float3                   cam_w;
  OptixTraversableHandle   handle;

  // Geometry arrays
  hipDeviceptr_t              d_vertices;   // float3*
  hipDeviceptr_t              d_indices;    // uint3*
  hipDeviceptr_t              d_normals;    // float3*
  uint32_t                 num_triangles;

  // Rectangular area light centered at light_pos with half extents light_half
  float3                   light_pos;
  float3                   light_emit;
  float3                   light_normal;
  float2                   light_half;
};

extern "C" {
__constant__ Params params;
}

// SBT data for each primitive
struct HitgroupData
{
  float3 kd; // diffuse reflectance
  float3 ke; // emission
};

enum {
    RAY_TYPE_RADIANCE = 0,
    RAY_TYPE_SHADOW   = 1,
    RAY_TYPE_COUNT    = 2
};

// --- float3 helpers (device-only) -----------------------------------------

static __forceinline__ __device__ float3 make3(float s) { return make_float3(s,s,s); }

static __forceinline__ __device__ float3 operator+(const float3& a, const float3& b) {
  return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}
static __forceinline__ __device__ float3 operator-(const float3& a, const float3& b) {
  return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}
static __forceinline__ __device__ float3 operator*(const float3& a, float s) {
  return make_float3(a.x*s, a.y*s, a.z*s);
}
static __forceinline__ __device__ float3 operator*(float s, const float3& a) {
  return make_float3(a.x*s, a.y*s, a.z*s);
}
static __forceinline__ __device__ float3 operator*(const float3& a, const float3& b) {
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}
static __forceinline__ __device__ float3 mul(const float3& a, const float3& b) {
  return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);
}
static __forceinline__ __device__ float3 operator/(const float3& a, float s) {
  float inv = 1.0f/s;
  return make_float3(a.x*inv, a.y*inv, a.z*inv);
}
static __forceinline__ __device__ float3& operator+=(float3& a, const float3& b) {
    a.x += b.x; a.y += b.y; a.z += b.z; return a;
}
static __forceinline__ __device__ float dot(const float3& a, const float3& b) {
    return a.x*b.x + a.y*b.y + a.z*b.z;
}
static __forceinline__ __device__ float3 cross(const float3& a, const float3& b) {
    return make_float3(a.y*b.z - a.z*b.y,
                       a.z*b.x - a.x*b.z,
                       a.x*b.y - a.y*b.x);
}
static __forceinline__ __device__ float length(const float3& v) {
    return sqrtf(dot(v, v));
}
static __forceinline__ __device__ float3 normalize(const float3& v) {
    const float len = length(v);
    return (len > 0.f) ? v / len : make_float3(0.f, 0.f, 0.f);
}
static __forceinline__ __device__ float  dot3(const float3& a, const float3& b) {
  return a.x*b.x + a.y*b.y + a.z*b.z;
}
static __forceinline__ __device__ float3 cross3(const float3& a, const float3& b) {
  return make_float3(a.y*b.z - a.z*b.y,
                     a.z*b.x - a.x*b.z,
                     a.x*b.y - a.y*b.x);
}
static __forceinline__ __device__ float  length3(const float3& a) {
  return sqrtf(dot3(a,a));
}
static __forceinline__ __device__ float3 normalize3(const float3& a) {
  return a / fmaxf(length3(a), 1e-20f);
}
static __forceinline__ __device__ float  clamp01(float x) {
  return fminf(fmaxf(x, 0.0f), 1.0f);
}
static __forceinline__ __device__ unsigned int pcg(unsigned long long& state) {
  unsigned long long oldstate = state;
  state = oldstate * 6364136223846793005ULL + 1442695040888963407ULL;
  unsigned int xorshifted = ((oldstate >> 18u) ^ oldstate) >> 27u;
  unsigned int rot = oldstate >> 59u;
  return (xorshifted >> rot) | (xorshifted << ((-rot) & 31));
}

static __forceinline__ __device__ float rnd(unsigned long long& state) {
  return float(pcg(state) >> 8) * (1.0f / float(0x01000000u));
}

static __forceinline__ __device__ void make_onb(const float3& n, float3& t, float3& b)
{
  if (n.z < -0.9999999f) {
    t = make_float3(0.0f, -1.0f, 0.0f);
    b = make_float3(-1.0f, 0.0f, 0.0f);
  } else {
    const float a = 1.0f / (1.0f + n.z);
    const float naxy = -n.x * n.y * a;
    t = make_float3(1.0f - n.x * n.x * a, naxy, -n.x);
    b = make_float3(naxy, 1.0f - n.y * n.y * a, -n.y);
  }
}

// --- payload packing -------------------------------------------------------

static __forceinline__ __device__ void packPtr(void* ptr, unsigned int& u0, unsigned int& u1) {
  uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
  u0 = static_cast<unsigned int>(uptr & 0xFFFFFFFFu);
  u1 = static_cast<unsigned int>(uptr >> 32);
}

template<typename T>
static __forceinline__ __device__ T* unpackPtr() {
  uint64_t u0 = static_cast<uint64_t>(optixGetPayload_0());
  uint64_t u1 = static_cast<uint64_t>(optixGetPayload_1());
  uint64_t uptr = (u1 << 32) | u0;
  return reinterpret_cast<T*>(uptr);
}

// --- per-ray data ----------------------------------------------------------

struct PRD {
    float3 radiance;
    float3 throughput;
    float3 origin;
    float3 direction;
    unsigned long long seed;
    int    depth;
    int    done;
    // MIS: previous bounce's BSDF pdf and validity flag
    float  prev_pdf_bsdf;
    int    prev_pdf_valid;
};

// --- access helpers for hipDeviceptr_t arrays --------------------------------

template<typename T>
static __forceinline__ __device__ T* ptr_at(hipDeviceptr_t p) {
  return reinterpret_cast<T*>(p);
}

// --- Miss programs ---------------------------------------------------------

extern "C" __global__ void __miss__ms_radiance()
{
  PRD* prd = unpackPtr<PRD>();
  prd->radiance = make3(0.0f);
  prd->done = 1;
}

extern "C" __global__ void __miss__ms_shadow()
{
  // 0 means not occluded; anyhit will set to 1
  optixSetPayload_0(0u);
}

// --- Anyhit for shadow rays: mark occluded and terminate -------------------

extern "C" __global__ void __anyhit__ah_shadow()
{
    optixSetPayload_0(1u);
    optixTerminateRay();
}

// --- Closest hit for radiance rays ----------------------------------------

extern "C" __global__ void __closesthit__ch()
{
    PRD& prd = *unpackPtr<PRD>();

    const unsigned int prim = optixGetPrimitiveIndex();

    // Geometry from Params (NOT from SBT)
    const uint3*  indices  = reinterpret_cast<const uint3*>(params.d_indices);
    const float3* vertices = reinterpret_cast<const float3*>(params.d_vertices);
    const uint3 tri = indices[prim];

    const float3 v0 = vertices[tri.x];
    const float3 v1 = vertices[tri.y];
    const float3 v2 = vertices[tri.z];

    const float2 bc = optixGetTriangleBarycentrics();
    const float b1 = bc.x;
    const float b2 = bc.y;
    const float b0 = 1.0f - b1 - b2;

    const float3 P  = v0 * b0 + v1 * b1 + v2 * b2;
    const float3 Ng = normalize3(reinterpret_cast<const float3*>(params.d_normals)[prim]);

    // Fetch per-triangle materials from SBT
    const HitgroupData* hg = reinterpret_cast<const HitgroupData*>(optixGetSbtDataPointer());
    const float3 kd = hg->kd;
    const float3 ke = hg->ke;

    // Sample direct illumination from rectangular area light
    unsigned long long& seed = prd.seed;
    float3 Lo = make3(0.0f);
    {
        float u = (rnd(seed) * 2.0f - 1.0f) * params.light_half.x;
        float v = (rnd(seed) * 2.0f - 1.0f) * params.light_half.y;
        float3 lp = make_float3(params.light_pos.x + u, params.light_pos.y, params.light_pos.z + v);
        float3 L = lp - P;
        float dist2 = fmaxf(dot(L, L), 1e-6f);
        float dist = sqrtf(dist2);
        float3 wi = L / dist;
        float3 light_n = make_float3(0.0f, -1.0f, 0.0f);
        float cosS = fmaxf(0.0f, dot(Ng, wi));
        float cosL = fmaxf(0.0f, dot(params.light_normal, wi * -1.0f));
        if (cosS > 0.0f && cosL > 0.0f) {
            unsigned int occluded = 0u;
            optixTrace(
                params.handle,
                P + Ng * 1e-3f,
                wi,
                0.0f,
                dist - 1e-3f,
                0.0f,
                1,
                OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
                RAY_TYPE_SHADOW,
                RAY_TYPE_COUNT,
                RAY_TYPE_SHADOW,
                occluded);
            if (!occluded) {
                float area = 4.0f * params.light_half.x * params.light_half.y;
                float pdf_area = 1.0f / area;
                float pdf_light = pdf_area * dist2 / cosL;
                float pdf_bsdf = cosS * (1.0f / HIP_PI_F);
                float w = pdf_light / (pdf_light + pdf_bsdf);
                float3 f = kd * (1.0f / HIP_PI_F);
                float3 contrib = params.light_emit * f * (cosS * cosL / dist2) / pdf_area;
                Lo += contrib * w;
            }
        }
    }

    // Accumulate emission with MIS and direct light
    float3 emission = ke;
    if (prd.prev_pdf_valid && (ke.x > 0.0f || ke.y > 0.0f || ke.z > 0.0f)) {
        float3 L = P - prd.origin;
        float dist2 = fmaxf(dot(L, L), 1e-6f);
        float area = 4.0f * params.light_half.x * params.light_half.y;
        float3 light_n = make_float3(0.0f, -1.0f, 0.0f);
        float cosL = fmaxf(0.0f, dot(params.light_normal, prd.direction * -1.0f));
        float pdf_light = dist2 / (cosL * area);
        float w_bsdf = prd.prev_pdf_bsdf / (prd.prev_pdf_bsdf + pdf_light);
        emission = emission * w_bsdf;
    }
    prd.radiance += emission + Lo;

    // Sample diffuse direction (cosine-weighted)
    float r1 = rnd(seed);
    float r2 = rnd(seed);
    const float phi = 2.0f * HIP_PI_F * r1;
    const float cosTheta = sqrtf(1.0f - r2);
    const float sinTheta = sqrtf(r2);
    float3 localDir = make_float3(cosf(phi) * sinTheta, sinf(phi) * sinTheta, cosTheta);

    float3 tangent, bitangent;
    make_onb(Ng, tangent, bitangent);
    float3 newDir = normalize(localDir.x * tangent + localDir.y * bitangent + localDir.z * Ng);

    prd.origin = P + Ng * 1e-3f;
    prd.direction = newDir;
    prd.throughput = mul(prd.throughput, kd);
    prd.prev_pdf_bsdf = cosTheta * (1.0f / HIP_PI_F);
    prd.prev_pdf_valid = 1;

    prd.depth++;
    if (prd.depth >= 5) {
        prd.done = 1;
        return;
    }
    if (prd.depth >= 3) {
        float p = fmaxf(prd.throughput.x, fmaxf(prd.throughput.y, prd.throughput.z));
        if (rnd(seed) > p) {
            prd.done = 1;
            return;
        }
        prd.throughput = prd.throughput / fmaxf(p, 1e-3f);
    }
}

// --- Raygen program --------------------------------------------------------

static __forceinline__ __device__ float3 sample_camera_dir(int x, int y, const float2& jitter)
{
  const uint3  dim = optixGetLaunchDimensions();
  const float  fx  = (float(x) + jitter.x) / float(dim.x);
  const float  fy  = (float(y) + jitter.y) / float(dim.y);
  const float2 d   = make_float2(2.0f*fx - 1.0f, 1.0f - 2.0f*fy);
  return normalize3(params.cam_w + d.x * params.cam_u + d.y * params.cam_v);
}

static __forceinline__ __device__ int bayer_channel_for(int x, int y, int pattern)
{
  const int mx = x & 1;
  const int my = y & 1;
  switch(pattern){
    case 0: return (my==0) ? (mx==0 ? 0:1) : (mx==0 ? 1:2);
    case 1: return (my==0) ? (mx==0 ? 2:1) : (mx==0 ? 1:0);
    case 2: return (my==0) ? (mx==0 ? 1:0) : (mx==0 ? 2:1);
    default:return (my==0) ? (mx==0 ? 1:2) : (mx==0 ? 0:1);
  }
}

extern "C" __global__ void __raygen__rg()
{
  const uint3  idx = optixGetLaunchIndex();
  const int    x   = int(idx.x);
  const int    y   = int(idx.y);
  const int    W   = params.width;
  const int    H   = params.height;
  const int    dst = y * W + x;

  // Trace radiance
  const float3 org = params.cam_eye;
  float3 sum_rgb = make3(0.0f);
  float sum_bayer = 0.0f;
  const int ch = bayer_channel_for(x, y, params.bayer_pattern);
  unsigned long long seed =
      ((unsigned long long)params.frame * 9781ULL) ^
      ((unsigned long long)dst * 6271ULL) ^
      0x853c49e6748fea9bULL;

  for (int s = 0; s < params.spp; ++s) {
    PRD prd;
    prd.radiance = make3(0.0f);
    prd.throughput = make3(1.0f);
    prd.origin = org;
    prd.direction = sample_camera_dir(x, y, make_float2(rnd(seed), rnd(seed)));
    prd.depth = 0;
    prd.done = 0;
    prd.seed = seed;
    prd.prev_pdf_bsdf = 0.0f;
    prd.prev_pdf_valid = 0;

    while (!prd.done) {
      prd.radiance = make3(0.0f);
      float3 throughput = prd.throughput;
	  float throughput_ch = (ch==0 ? throughput.x : (ch==1 ? throughput.y : throughput.z));
      unsigned int u0, u1; packPtr(&prd, u0, u1);
      optixTrace(
        params.handle,
        prd.origin,
        prd.direction,
        0.0f, 1e16f, 0.0f,
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_DISABLE_ANYHIT,
        0,
        2,
        0,
        u0, u1);
      if (params.out_rgb)
        sum_rgb += prd.radiance * throughput;
      if (params.out_bayer)
        sum_bayer += (ch==0 ? prd.radiance.x : (ch==1 ? prd.radiance.y : prd.radiance.z)) * throughput_ch;
    }

    seed = prd.seed;
  }

  // Write outputs
  if (params.out_rgb) {
		float3 rad = sum_rgb / float(params.spp);
    float3* out = ptr_at<float3>(params.out_rgb);
    out[dst] = rad;
  }
  if (params.out_bayer) {
		float rad = fmaxf(sum_bayer / float(params.spp), 0.f);
    float* out = ptr_at<float>(params.out_bayer);
    out[dst] = rad;
  }
}
